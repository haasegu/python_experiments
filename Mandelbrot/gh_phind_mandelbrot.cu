#include "hip/hip_runtime.h"
// /usr/local/cuda/bin/nvcc  -g -pg -Xcompiler -fmax-errors=1 -Xcompiler -O3 -std=c++20 --expt-relaxed-constexpr -O3 -use_fast_math -restrict --ftemplate-backtrace-limit 1 -gencode arch=compute_75,code=\"compute_75,sm_75\" -gencode arch=compute_80,code=\"compute_80,sm_80\"  --ptxas-options=-v,-warn-spills --resource-usage -src-in-ptx --restrict --Wreorder --ftemplate-backtrace-limit 1 -res-usage -Wno-deprecated-declarations  --compiler-options=-fopenmp,-O3,-funsafe-math-optimizations  gh_phind_mandelbrot.cu

#include "timing.h"    // GH
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define BLOCK_SIZE 16
#define MAX_ITERATIONS 1000

__global__ void mandelbrotKernel(int width, int height, int max_iterations, double *result) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    double c_x = (x / (double)width) * 3.5f - 2.5f;
    double c_y = (y / (double)height) * 2.0f - 1.0f;
    
    double z_x = 0.0f;
    double z_y = 0.0f;
    int iter = 0; // GH
    for (iter = 0; iter < max_iterations; iter++) { // GH
    //for (int iter = 0; iter < max_iterations; iter++) {
        double temp = z_x * z_x - z_y * z_y + c_x;
        z_y = 2.0f * z_x * z_y + c_y;
        z_x = temp;
        
        if (z_x * z_x + z_y * z_y > 4.0f*4.0f) break;
    }
    
    result[y * width + x] = iter;
}

int main() {
    const int WIDTH = 1920;
    const int HEIGHT = 1080;
    
    // Host variables
    double *h_result;
    hipHostMalloc(&h_result, WIDTH * HEIGHT * sizeof(double));
    
    // Device variables
    double *d_result;
    hipMalloc(&d_result, WIDTH * HEIGHT * sizeof(double));
    
    // Kernel launch configuration
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((WIDTH + blockSize.x - 1) / blockSize.x, 
                    (HEIGHT + blockSize.y - 1) / blockSize.y);
    
    tic();  // GH
    // Launch kernel
    mandelbrotKernel<<<gridSize, blockSize>>>(WIDTH, HEIGHT, MAX_ITERATIONS, d_result);
    
    // Copy result from device to host
    hipMemcpy(h_result, d_result, WIDTH * HEIGHT * sizeof(double), hipMemcpyDeviceToHost);
    double tdiff = toc();  // Gh
    
    // Print result
    for (int y = 0; y < HEIGHT; y++) {
        for (int x = 0; x < WIDTH; x++) {
            printf("Result[%d,%d] = %d\n", x, y, (int)h_result[y * WIDTH + x]);
        }
    }
    
    // Free device memory
    hipFree(d_result);
    
    // Free host memory
    hipHostFree(h_result);
    
    printf("Run time: %f\n",tdiff);  // GH
    
    return 0;
}
